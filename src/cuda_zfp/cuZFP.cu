#include "hip/hip_runtime.h"
#include <assert.h>

#include "cuZFP.h"

#include "encode1.cuh"
#include "encode2.cuh"
#include "encode3.cuh"

#include "decode1.cuh"
#include "decode2.cuh"
#include "decode3.cuh"

#include "ErrorCheck.h"

#include "pointers.cuh"
#include "type_info.cuh"
#include <iostream>
#include <assert.h>

// we need to know about bitstream, but we don't 
// want duplicate symbols.
#ifndef inline_
  #define inline_ inline
#endif

#include "zfp/bitstream.inl"
namespace internal 
{ 
  
bool is_contigous3d(const uint dims[3], const int3 &stride, long long int &offset)
{
  typedef long long int int64;
  int64 idims[3];
  idims[0] = dims[0];
  idims[1] = dims[1];
  idims[2] = dims[2];

  int64 imin = std::min(stride.x,0) * (idims[0] - 1) + 
               std::min(stride.y,0) * (idims[1] - 1) + 
               std::min(stride.z,0) * (idims[2] - 1);

  int64 imax = std::max(stride.x,0) * (idims[0] - 1) + 
               std::max(stride.y,0) * (idims[1] - 1) + 
               std::max(stride.z,0) * (idims[2] - 1);
  offset = imin;
  int64 ns = idims[0] * idims[1] * idims[2];

  return (imax - imin + 1 == ns);
}

bool is_contigous2d(const uint dims[3], const int3 &stride, long long int &offset)
{
  typedef long long int int64;
  int64 idims[2];
  idims[0] = dims[0];
  idims[1] = dims[1];

  int64 imin = std::min(stride.x,0) * (idims[0] - 1) + 
               std::min(stride.y,0) * (idims[1] - 1);

  int64  imax = std::max(stride.x,0) * (idims[0] - 1) + 
                std::max(stride.y,0) * (idims[1] - 1); 

  offset = imin;
  return (imax - imin + 1) == (idims[0] * idims[1]);
}

bool is_contigous1d(uint dim, const int &stride, long long int &offset)
{
  offset = 0;
  if(stride < 0) offset = stride * (int(dim) - 1);
  return std::abs(stride) == 1;
}

bool is_contigous(const uint dims[3], const int3 &stride, long long int &offset)
{
  int d = 0;
  
  if(dims[0] != 0) d++;
  if(dims[1] != 0) d++;
  if(dims[2] != 0) d++;

  if(d == 3)
  {
    return is_contigous3d(dims, stride, offset);
  }
  else if(d == 2)
  {
   return is_contigous2d(dims, stride, offset);
  }
  else
  {
    return is_contigous1d(dims[0], stride.x, offset);
  } 

}
//
// encode expects device pointers
//
template<typename T>
size_t encode(uint dims[3], int3 stride, int bits_per_block, T *d_data, Word *d_stream, hipStream_t custream)
{

  int d = 0;
  size_t len = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      d++;
      len *= dims[i];
    }
  }

  ErrorCheck errors;
  size_t stream_size = 0;
  if(d == 1)
  {
    int dim = dims[0];
    int sx = stride.x;
    stream_size = cuZFP::encode1<T>(dim, sx, d_data, d_stream, bits_per_block, custream); 
  }
  else if(d == 2)
  {
    uint2 ndims = make_uint2(dims[0], dims[1]);
    int2 s;
    s.x = stride.x; 
    s.y = stride.y; 
    stream_size = cuZFP::encode2<T>(ndims, s, d_data, d_stream, bits_per_block, custream); 
  }
  else if(d == 3)
  {
    int3 s;
    s.x = stride.x; 
    s.y = stride.y; 
    s.z = stride.z; 
    uint3 ndims = make_uint3(dims[0], dims[1], dims[2]);
    stream_size = cuZFP::encode3<T>(ndims, s, d_data, d_stream, bits_per_block, custream); 
  }

  errors.chk("Encode");
  
  return stream_size; 
}

template<typename T>
size_t decode(uint ndims[3], int3 stride, int bits_per_block, Word *stream, T *out, hipStream_t custream)
{

  /* Include CUDA stream in decode call */

  int d = 0;
  size_t out_size = 1;
  size_t stream_bytes = 0;
  for(int i = 0; i < 3; ++i)
  {
    if(ndims[i] != 0)
    {
      d++;
      out_size *= ndims[i];
    }
  }

  if(d == 3)
  {
    uint3 dims = make_uint3(ndims[0], ndims[1], ndims[2]);

    int3 s;
    s.x = stride.x; 
    s.y = stride.y; 
    s.z = stride.z; 

    stream_bytes = cuZFP::decode3<T>(dims, s, stream, out, bits_per_block, custream); 
  }
  else if(d == 1)
  {
    uint dim = ndims[0];
    int sx = stride.x;

    stream_bytes = cuZFP::decode1<T>(dim, sx, stream, out, bits_per_block, custream); 

  }
  else if(d == 2)
  {
    uint2 dims;
    dims.x = ndims[0];
    dims.y = ndims[1];

    int2 s;
    s.x = stride.x; 
    s.y = stride.y; 

    stream_bytes = cuZFP::decode2<T>(dims, s, stream, out, bits_per_block, custream); 
  }
  else std::cerr<<" d ==  "<<d<<" not implemented\n";
 
  return stream_bytes;
}

Word *setup_device_stream_compress(zfp_stream *stream,const zfp_field *field)
{
  bool stream_device = cuZFP::is_gpu_ptr(stream->stream->begin);
  assert(sizeof(bitstream_word) == sizeof(Word)); // "CUDA version currently only supports 64bit words");

  if(stream_device)
  {
    return (Word*) stream->stream->begin;
  }

  Word *d_stream = NULL;
  size_t max_size = zfp_stream_maximum_size(stream, field);
  hipMallocAsync(&d_stream, max_size, field->cuStream);
  return d_stream;
}

Word *setup_device_stream_decompress(zfp_stream *stream,const zfp_field *field)
{
  bool stream_device = cuZFP::is_gpu_ptr(stream->stream->begin);
  assert(sizeof(bitstream_word) == sizeof(Word)); // "CUDA version currently only supports 64bit words");

  if(stream_device)
  {
    return (Word*) stream->stream->begin;
  }

  Word *d_stream = NULL;
  //TODO: change maximum_size to compressed stream size
  size_t size = zfp_stream_maximum_size(stream, field);

  /* Allocate memory per CUDA stream */

  hipMallocAsync(&d_stream, size, field->cuStream);
  hipMemcpyAsync(d_stream, stream->stream->begin, size, hipMemcpyHostToDevice, field->cuStream);
  return d_stream;
}

void * offset_void(zfp_type type, void *ptr, long long int offset)
{
  void * offset_ptr = NULL;
  if(type == zfp_type_float)
  {
    float* data = (float*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  else if(type == zfp_type_double)
  {
    double* data = (double*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  else if(type == zfp_type_int32)
  {
    int * data = (int*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  else if(type == zfp_type_int64)
  {
    long long int * data = (long long int*) ptr;
    offset_ptr = (void*)(&data[offset]);
  }
  return offset_ptr;
}

void *setup_device_field_compress(const zfp_field *field, const int3 &stride, long long int &offset)
{
  bool field_device = cuZFP::is_gpu_ptr(field->data);

  if(field_device)
  {
    offset = 0;
    return field->data;
  }
  
  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;

  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }

  bool contig = internal::is_contigous(dims, stride, offset);
  
  void * host_ptr = offset_void(field->type, field->data, offset);;

  void *d_data = NULL;
  if(contig)
  {
    size_t field_bytes = type_size * field_size;
    /* allocate memory async per stream */

    hipMallocAsync(&d_data, field_bytes, field->cuStream);

    hipMemcpyAsync(d_data, host_ptr, field_bytes, hipMemcpyHostToDevice, field->cuStream);
  }
  return offset_void(field->type, d_data, -offset);
}

void *setup_device_field_decompress(const zfp_field *field, const int3 &stride, long long int &offset)
{
  bool field_device = cuZFP::is_gpu_ptr(field->data);

  if(field_device)
  {
    offset = 0;
    return field->data;
  }

  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;

  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }

  bool contig = internal::is_contigous(dims, stride, offset);

  void *d_data = NULL;
  if(contig)
  {
    size_t field_bytes = type_size * field_size;
    /* Allocate GPU memory per CUDA stream */

    hipMallocAsync(&d_data, field_bytes, field->cuStream);
  }
  return offset_void(field->type, d_data, -offset);
}

/* CUDA stream is assigned in the device cleanup */

void cleanup_device_ptr(void *orig_ptr, void *d_ptr, size_t bytes, long long int offset, zfp_type type, hipStream_t custream)
{
  bool device = cuZFP::is_gpu_ptr(orig_ptr);
  if(device)
  {
    return;
  }
  // from whence it came
  void *d_offset_ptr = offset_void(type, d_ptr, offset);
  void *h_offset_ptr = offset_void(type, orig_ptr, offset);

  if(bytes > 0)
  {
    hipMemcpyAsync(h_offset_ptr, d_offset_ptr, bytes, hipMemcpyDeviceToHost, custream);
  }

  hipFreeAsync(d_offset_ptr, custream);
}

} // namespace internal

size_t
cuda_compress(zfp_stream *stream, const zfp_field *field)
{
  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;

  int3 stride;  
  stride.x = field->sx ? field->sx : 1;
  stride.y = field->sy ? field->sy : field->nx;
  stride.z = field->sz ? field->sz : field->nx * field->ny;

  /* CUDA stream implementation */

  hipStream_t cudastream = field->cuStream;
  
  size_t stream_bytes = 0;
  long long int offset = 0; 
  void *d_data = internal::setup_device_field_compress(field, stride, offset);

  if(d_data == NULL)
  {
    // null means the array is non-contiguous host mem which is not supported
    return 0;
  }

  Word *d_stream = internal::setup_device_stream_compress(stream, field);

  if(field->type == zfp_type_float)
  {
    float* data = (float*) d_data;
    stream_bytes = internal::encode<float>(dims, stride, (int)stream->maxbits, data, d_stream, cudastream);
  }
  else if(field->type == zfp_type_double)
  {
    double* data = (double*) d_data;
    stream_bytes = internal::encode<double>(dims, stride, (int)stream->maxbits, data, d_stream, cudastream);
  }
  else if(field->type == zfp_type_int32)
  {
    int * data = (int*) d_data;
    stream_bytes = internal::encode<int>(dims, stride, (int)stream->maxbits, data, d_stream, cudastream);
  }
  else if(field->type == zfp_type_int64)
  {
    long long int * data = (long long int*) d_data;
    stream_bytes = internal::encode<long long int>(dims, stride, (int)stream->maxbits, data, d_stream, cudastream);
  }

  internal::cleanup_device_ptr(stream->stream->begin, d_stream, stream_bytes, 0, field->type, cudastream);
  internal::cleanup_device_ptr(field->data, d_data, 0, offset, field->type, cudastream);

  // zfp wants to flush the stream.
  // set bits to wsize because we already did that.
  size_t compressed_size = stream_bytes / sizeof(Word);
  stream->stream->bits = wsize;
  // set stream pointer to end of stream
  stream->stream->ptr = stream->stream->begin + compressed_size;

  return stream_bytes;
}
  
void 
cuda_decompress(zfp_stream *stream, zfp_field *field)
{
  uint dims[3];
  dims[0] = field->nx;
  dims[1] = field->ny;
  dims[2] = field->nz;
   
  int3 stride;  
  stride.x = field->sx ? field->sx : 1;
  stride.y = field->sy ? field->sy : field->nx;
  stride.z = field->sz ? field->sz : field->nx * field->ny;

  size_t decoded_bytes = 0;
  long long int offset = 0;
  void *d_data = internal::setup_device_field_decompress(field, stride, offset);
  
  if(d_data == NULL)
  {
    // null means the array is non-contiguous host mem which is not supported
    return;
  }

  /* Include CUDA Stream */

  hipStream_t cudastream = field->cuStream;

  Word *d_stream = internal::setup_device_stream_decompress(stream, field);

  if(field->type == zfp_type_float)
  {
    float *data = (float*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data, cudastream);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_double)
  {
    double *data = (double*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data, cudastream);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_int32)
  {
    int *data = (int*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data, cudastream);
    d_data = (void*) data;
  }
  else if(field->type == zfp_type_int64)
  {
    long long int *data = (long long int*) d_data;
    decoded_bytes = internal::decode(dims, stride, (int)stream->maxbits, d_stream, data, cudastream);
    d_data = (void*) data;
  }
  else
  {
    std::cerr<<"Cannot decompress: type unknown\n";
  }

   
  size_t type_size = zfp_type_size(field->type);

  size_t field_size = 1;
  for(int i = 0; i < 3; ++i)
  {
    if(dims[i] != 0)
    {
      field_size *= dims[i];
    }
  }
  
  size_t bytes = type_size * field_size;
  internal::cleanup_device_ptr(stream->stream->begin, d_stream, 0, 0, field->type, cudastream);
  internal::cleanup_device_ptr(field->data, d_data, bytes, offset, field->type, cudastream);
  
  // this is how zfp determines if this was a success
  size_t words_read = decoded_bytes / sizeof(Word);
  stream->stream->bits = wsize;
  // set stream pointer to end of stream
  stream->stream->ptr = stream->stream->begin + words_read;
}
